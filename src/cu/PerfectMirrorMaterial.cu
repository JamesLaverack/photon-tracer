#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <hiprand/hiprand_kernel.h>
#include "PerRay.h"

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  photon_ray_type, , );
rtDeclareVariable(unsigned int,  scene_bounce_limit, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint,      launch_index, rtLaunchIndex, );
rtBuffer<hiprandState, 1>              states;

// Photon ray datatype
rtDeclareVariable(PerRayData_photon, prd_photon, rtPayload, );
rtDeclareVariable(int, follow_photon, , );

// Current Ray & Intersection
rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );

// Object spesific
RT_PROGRAM void closest_hit() {
	// Do we absorb this?
	if(prd_photon.depth >= scene_bounce_limit) return;
	float const pi = 3.141;

	float3 i = ray.direction*-1;
	float m = optix::dot(i, geometric_normal)*2;
	float3 reflect = optix::normalize(geometric_normal*m-i);
	// Get needed values
	float3 hitpoint = ray.origin + t_hit * ray.direction;
	// Fire new ray!
	optix::Ray new_ray = optix::make_Ray(hitpoint, reflect, photon_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_photon prd_bounce;
	prd_bounce.importance = 1.f;
	prd_bounce.depth = prd_photon.depth+1;
	prd_bounce.wavelength = prd_photon.wavelength;
	rtTrace(top_object, new_ray, prd_bounce);
}
