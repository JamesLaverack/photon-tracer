#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <hiprand/hiprand_kernel.h>
#include "PerRay.h"

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  photon_ray_type, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  iterations, , );
rtDeclareVariable(uint,      launch_index, rtLaunchIndex, );

// Object spesific
rtDeclareVariable(float3,            location, , );
rtDeclareVariable(float3,            normal, , );
rtDeclareVariable(float3,            up, , );
rtDeclareVariable(float3,            right, , );
rtDeclareVariable(float,            width, , );
rtDeclareVariable(float,            height, , );
rtDeclareVariable(float,            variance, , );
rtBuffer<hiprandState, 1>              states;

RT_PROGRAM void light() {
	int report = 0;
	if(launch_index == 0) report = 1;

	for(int i=0;i<iterations;i++) {
		float3 pos = location;
		pos += up*hiprand_uniform(&states[launch_index])*height;
		pos += right*hiprand_uniform(&states[launch_index])*width;
	
		float4 ray_direction = make_float4(normal);
		float phi = hiprand_uniform(&states[launch_index])*variance*2 - variance;
		float theta = hiprand_uniform(&states[launch_index])*variance*2 - variance;
		optix::Matrix4x4 rot1 = optix::Matrix4x4::rotate(phi  , up);
		ray_direction = ray_direction*rot1;
		optix::Matrix4x4 rot2 = optix::Matrix4x4::rotate(theta, right);
		ray_direction = ray_direction*rot2;
		
		optix::Ray ray = optix::make_Ray(pos, make_float3(ray_direction), photon_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		PerRayData_photon prd;
		prd.importance = 1.f;
		prd.depth = 0;
		prd.wavelength = hiprand_uniform(&states[launch_index])*400+380;
		rtTrace(top_object, ray, prd);
	}
	if(report) rtPrintf("Ran all iterations.\n");
}
