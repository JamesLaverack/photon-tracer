#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <hiprand/hiprand_kernel.h>
#include "PerRay.h"

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  photon_ray_type, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  iterations, , );
rtDeclareVariable(uint,      launch_index, rtLaunchIndex, );

// Object spesific
rtDeclareVariable(float3,            location, , );
rtBuffer<hiprandState, 1>              states;

RT_PROGRAM void light() {
	// Point light, so always emmit from 1 place
	//printf("HURF\n");
	//r->getPosition().print();
	//r->getPosition().print();
	//printf("DURF\n");
	// Randomise direction
	//TODO Frame number?
	/* Copy state to local memory for efficiency */
	int report = 0;
	if(launch_index == 0) report = 1;

	for(int i=0;i<iterations;i++) {
		float a = hiprand_uniform(&states[launch_index]);
		float b = hiprand_uniform(&states[launch_index]);
		float phi = 2*3.141*a;
		float theta = std::acos(2*b-1);
// 		rtPrintf("theta %f phi %f\n", theta, phi);
		float x, y, z;
		// Rotate<1, 0, 0> by theta around the y axis
		x = (float) std::sin(theta)*std::cos(phi);
		z = (float) std::sin(theta)*std::sin(phi);
		y = (float) std::cos(theta);

		float3 ray_direction = make_float3(x, y, z);
		
		optix::Ray ray = optix::make_Ray(location, ray_direction, photon_ray_type, scene_epsilon, RT_DEFAULT_MAX);

		PerRayData_photon prd;
		prd.importance = 1.f;
		prd.depth = 0;
		prd.wavelength = hiprand_uniform(&states[launch_index])*400+300;
		rtTrace(top_object, ray, prd);
	}
	if(report) rtPrintf("Ran all iterations.\n");
}
