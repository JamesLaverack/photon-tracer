#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <hiprand/hiprand_kernel.h>
#include "NormalRandom.h"
#include "PerRay.h"

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  photon_ray_type, , );
rtDeclareVariable(unsigned int,  scene_bounce_limit, , );
rtDeclareVariable(rtObject,      top_object, , );

// Photon ray datatype
rtDeclareVariable(PerRayData_photon, prd_photon, rtPayload, );

// Current Ray & Intersection
rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 


// Object spesific
rtDeclareVariable(float, max_wavelength, , );
rtDeclareVariable(float, min_wavelength, , );
rtDeclareVariable(float, standard_deviation, , );

RT_PROGRAM void closest_hit() {
	// Do we absorb this?
	if(prd_photon.wavelength>max_wavelength) return;
	if(prd_photon.wavelength<min_wavelength) return;
	if(prd_photon.depth >= scene_bounce_limit) return;

	float3 reverse_normal = (geometric_normal)*-1;
	float reflect_angle = std::acos(optix::dot(ray.direction, reverse_normal));
	// project our incident ray onto the plane defined by
	// < hitLocation, normal > and make sure it's a unit vector, this becomes u.
	float m = optix::dot(geometric_normal, ray.direction);
	float3 u_vec = optix::normalize( (ray.direction)-((geometric_normal)*m) );
	// Calculate v from the cross product of u and normal
	float3 v_vec = optix::normalize( optix::cross( u_vec, geometric_normal) );
	reflect_angle = -reflect_angle;
	// get theta, which is the angle between our bounce and the normal in the u direction.
	// Also get phi, the angle between our bounce and the normal in the v direction.

	float theta = reflect_angle+hiprand_normal(&prd_photon.rand_state)*standard_deviation;
	float phi   = 0+hiprand_normal(&prd_photon.rand_state)*standard_deviation;

	// Construct our bounce vector, this is our actual reflection.
	float4 bounce = optix::make_float4( geometric_normal.x, geometric_normal.y, geometric_normal.z, 0);
	// Do some rotation
	optix::Matrix4x4 rot1 = optix::Matrix4x4::rotate(theta, v_vec);
	bounce = bounce*rot1;
	optix::Matrix4x4 rot2 = optix::Matrix4x4::rotate(phi  , u_vec);
	bounce = bounce*rot2;
	// Get needed values
	float3 bounce_direction = optix::normalize( optix::make_float3(bounce.x, bounce.y, bounce.z) );
	float3 hitpoint = ray.origin + t_hit * ray.direction;
	// Fire new ray!
	optix::Ray new_ray = optix::make_Ray(hitpoint, bounce_direction, photon_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_photon prd_bounce;
	prd_bounce.importance = 1.f;
	prd_bounce.depth = prd_photon.depth+1;
	prd_bounce.rand_state = prd_photon.rand_state;
	prd_bounce.wavelength = prd_photon.wavelength;
	rtTrace(top_object, new_ray, prd_bounce);
	prd_photon.rand_state = prd_bounce.rand_state;
}